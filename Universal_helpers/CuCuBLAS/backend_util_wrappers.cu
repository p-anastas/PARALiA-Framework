///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

/*void print_devices() {
  hipDeviceProp_t properties;
  int nDevices = 0;
  massert(HIPBLAS_STATUS_SUCCESS == hipGetDeviceCount(&nDevices), "print_devices: hipGetDeviceCount failed");
  printf("Found %d Devices: \n\n", nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&properties, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", properties.name);
    printf("  Memory Clock Rate (MHz): %d\n",
           properties.memoryClockRate / 1024);
    printf("  Memory Bus Width (bits): %d\n", properties.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0 * properties.memoryClockRate * (properties.memoryBusWidth / 8) /
               1.0e6);
    if (properties.major >= 3)
      printf("  Unified Memory support: YES\n\n");
    else
      printf("  Unified Memory support: NO\n\n");
  }
}
*/

void CoCoSyncCheckErr(){
  hipError_t errSync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
}

void CoCoASyncCheckErr(){
  hipError_t errAsync = hipGetLastError();
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}

void cudaCheckErrors(){
	//CoCoASyncCheckErr();
	CoCoSyncCheckErr();
}

int CoCoPeLiaGetDevice(){
  int dev_id = -1;
  hipError_t err = hipGetDevice(&dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaGetDevice: hipGetDevice failed - %s\n", hipGetErrorString(err));
  return dev_id;
}

void CoCoPeLiaSelectDevice(short dev_id){
  int dev_count;
  hipError_t err = hipGetDeviceCount(&dev_count);
  if(dev_id >= 0 && dev_id < dev_count){
  hipError_t err = hipSetDevice(dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaSelectDevice(%d): hipSetDevice(%d) failed - %s\n", dev_id, dev_id, hipGetErrorString(err));
  }
  else if(dev_id == -1){  /// "Host" device loc id used by CoCoPeLia
    hipSetDevice(0);
  }
  else error("CoCoPeLiaSelectDevice(%d): invalid dev_id\n", dev_id);
}

void CoCoPeLiaDevGetMemInfo(long long* free_dev_mem, long long* max_dev_mem){
  size_t free_dev_mem_tmp, max_dev_mem_tmp;
    int tmp_dev_id;
    hipError_t err = hipGetDevice(&tmp_dev_id);
    // TODO: For the CPU this function returns device 0 memory availability. Its a feature not a bug.
    massert(hipSuccess == err,
      "CoCoPeLiaDevGetMemInfo: hipGetDevice failed - %s\n", hipGetErrorString(err));
    err = hipMemGetInfo(&free_dev_mem_tmp, &max_dev_mem_tmp);
  	massert(hipSuccess == err,
      "CoCoPeLiaDevGetMemInfo: hipMemGetInfo failed - %s\n", hipGetErrorString(err));
    *free_dev_mem = (long long) free_dev_mem_tmp;
    *max_dev_mem = (long long) max_dev_mem_tmp;
}
