///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Backened wrapped host functions for cuda queue firing -> void func (void*)
///

#include <cblas.h>

#include "backend_wrappers.hpp"

void CoCoQueueLock(void* wrapped_lock){
#ifdef ENABLE_MUTEX_LOCKING
  (*(std::mutex*)wrapped_lock).lock();
#else
  while(__sync_lock_test_and_set ((&(*((int*)wrapped_lock))), 1));
#endif
#ifdef DEBUG
  lprintf(6, "CoCoQueueLock(%p) ran succesfully.\n", wrapped_lock);
#endif
}

void CoCoQueueUnlock(void* wrapped_lock){
#ifdef ENABLE_MUTEX_LOCKING
	(*(std::mutex*)wrapped_lock).unlock();
#else
  //int* intptr = (int*) wrapped_lock;
  //*intptr = 0;
  __sync_lock_release((&(*((int*) wrapped_lock))));
#endif

#ifdef DEBUG
  lprintf(6, "CoCoQueueUnlock(%p) ran succesfully.\n", wrapped_lock);
#endif
}

void CoCoIncAsync(void* wrapped_ptr_int){
  Ptr_atomic_int_p unwrapped = (Ptr_atomic_int_p) wrapped_ptr_int;
  *(unwrapped->ato_int_ptr)++;
  free(unwrapped);
#ifdef DEBUG
  lprintf(6, "CoCoIncAsync(%p, new_val=%d) ran succesfully.\n", unwrapped->ato_int_ptr, (*(unwrapped->ato_int_ptr)).load());
#endif
}

void CoCoDecAsync(void* wrapped_ptr_int){
  Ptr_atomic_int_p unwrapped = (Ptr_atomic_int_p) wrapped_ptr_int;
  (*(unwrapped->ato_int_ptr))--;
  free(unwrapped);
#ifdef DEBUG
  lprintf(6, "CoCoDecAsync(%p, new_val=%d) ran succesfully.\n", unwrapped->ato_int_ptr, (*(unwrapped->ato_int_ptr)).load());
#endif
}

void CoCoSetInt(void* wrapped_ptr_and_val){
  Ptr_and_int_p unwrapped = (Ptr_and_int_p) wrapped_ptr_and_val;
  *(unwrapped->int_ptr) = unwrapped->val;
  free(unwrapped);
#ifdef DEBUG
  lprintf(6, "CoCoSetVal(%p, %d) ran succesfully.\n", unwrapped->int_ptr, unwrapped->val);
#endif
}

void CoCoSetPtr(void* wrapped_ptr_and_parent){
  Ptr_and_parent_p unwrapped = (Ptr_and_parent_p) wrapped_ptr_and_parent;
  void* prev_ptr = *(unwrapped->ptr_parent);
  *(unwrapped->ptr_parent) = unwrapped->ptr_val;
  free(unwrapped);
#ifdef DEBUG
  lprintf(6, "CoCoSetPtr(prev=%p, %p) ran succesfully.\n", prev_ptr, unwrapped->ptr_val);
#endif
}

void CoCoSetTimerAsync(void* wrapped_timer_Ptr){
  double* timer = (double*) wrapped_timer_Ptr;
  *timer = csecond();
#ifdef DEBUG
  lprintf(6, "CoCoSetTimerAsync(%p) ran succesfully.\n", wrapped_timer_Ptr);
#endif
}

void cblas_wrap_daxpy(void* backend_data){
  axpy_backend_in_p ptr_ker_translate = (axpy_backend_in_p) backend_data;
  cblas_daxpy(ptr_ker_translate->N, ptr_ker_translate->alpha,
    (double*) *ptr_ker_translate->x, ptr_ker_translate->incx, (double*)
    *ptr_ker_translate->y, ptr_ker_translate->incy);
}

void cblas_wrap_saxpy(void* backend_data){
  axpy_backend_in_p ptr_ker_translate = (axpy_backend_in_p) backend_data;
  cblas_saxpy(ptr_ker_translate->N, ptr_ker_translate->alpha,
    (float*) *ptr_ker_translate->x, ptr_ker_translate->incx, (float*)
    *ptr_ker_translate->y, ptr_ker_translate->incy);
}

void cblas_wrap_dgemm(void* backend_data){
  short lvl = 6;
  gemm_backend_in_p ptr_ker_translate = (gemm_backend_in_p) backend_data;
#ifdef DDEBUG
  if (ptr_ker_translate->dev_id != -1)
    warning("cblas_wrap_dgemm: Suspicious device %d instead of -1\n", ptr_ker_translate->dev_id);
#endif
#ifdef DDEBUG
  lprintf(lvl, "cblas_wrap_dgemm: cblas_dgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif
  cblas_dgemm(CblasColMajor,
    OpCharToCblas(ptr_ker_translate->TransA), OpCharToCblas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
}

void cblas_wrap_sgemm(void* backend_data){
error("cblas_wrap_sgemm: never let empty unimplimented wrapped functions, moron\n");
}

void cublas_wrap_daxpy(void* backend_data, void* queue_wrap_p){
  axpy_backend_in_p ptr_ker_translate = (axpy_backend_in_p) backend_data;
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);
#ifdef ENABLE_PARALLEL_BACKEND
  hipblasHandle_t temp_handle = *((hipblasHandle_t*)((CQueue_p)queue_wrap_p)->cqueue_backend_data
    [((CQueue_p)queue_wrap_p)->backend_ctr]);
#else
  hipblasHandle_t temp_handle = *((hipblasHandle_t*)((CQueue_p)queue_wrap_p)->cqueue_backend_data);
#endif
  massert(HIPBLAS_STATUS_SUCCESS == hipblasDaxpy(temp_handle,
    ptr_ker_translate->N, (double*) &ptr_ker_translate->alpha, (double*) *ptr_ker_translate->x,
    ptr_ker_translate->incx, (double*) *ptr_ker_translate->y, ptr_ker_translate->incy),
    "cublas_wrap_daxpy failed\n");
}

void cublas_wrap_dgemm(void* backend_data, void* queue_wrap_p){
  short lvl = 6;
  gemm_backend_in_p ptr_ker_translate = (gemm_backend_in_p) backend_data;
#ifdef DDEBUG
  int cur_dev_id = CoCoPeLiaGetDevice();
  if (ptr_ker_translate->dev_id != cur_dev_id)
    warning("cublas_wrap_dgemm: Changing device %d -> %d\n", cur_dev_id, ptr_ker_translate->dev_id);
#endif
  CoCoPeLiaSelectDevice(ptr_ker_translate->dev_id);
#ifdef DDEBUG
  lprintf(lvl, "cublas_wrap_dgemm: hipblasDgemm(dev_id = %d, TransA = %c, TransB = %c,\
    M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
    B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n",
    ptr_ker_translate->dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha,
    (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
#endif
#ifdef ENABLE_PARALLEL_BACKEND
  hipblasHandle_t temp_handle = *((hipblasHandle_t*)((CQueue_p)queue_wrap_p)->cqueue_backend_data
    [((CQueue_p)queue_wrap_p)->backend_ctr]);
#else
  hipblasHandle_t temp_handle = *((hipblasHandle_t*)((CQueue_p)queue_wrap_p)->cqueue_backend_data);
#endif
  massert(HIPBLAS_STATUS_SUCCESS == hipblasDgemm(temp_handle,
    OpCharToCublas(ptr_ker_translate->TransA), OpCharToCublas(ptr_ker_translate->TransB),
    ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, &ptr_ker_translate->alpha,
    (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
    (VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB,
    &ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC),
    "cublas_wrap_dgemm: hipblasDgemm failed\n");
}
