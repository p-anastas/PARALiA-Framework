///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

int lvl = 1;

int Event_num_device[128] = {0};
#ifndef UNIHELPER_LOCKFREE_ENABLE
int unihelper_lock = 0;
#endif

inline void get_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	while(__sync_lock_test_and_set (&unihelper_lock, 1));
#endif
	;
}
inline void release_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	__sync_lock_release(&unihelper_lock);
#endif
	;
}

/*****************************************************/
/// Event Status-related functions

const char* print_event_status(event_status in_status){
	switch(in_status){
		case(UNRECORDED):
			return "UNRECORDED";
		case(RECORDED):
			return "RECORDED";
		case(COMPLETE):
			return "COMPLETE";
		case(CHECKED):
			return "CHECKED";
		case(GHOST):
			return "GHOST";
		default:
			error("print_event_status: Unknown state\n");
	}
}

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue(int dev_id_in)
{
	int prev_dev_id = CoCoPeLiaGetDevice();
	dev_id = dev_id_in;
	CoCoPeLiaSelectDevice(dev_id);
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::CommandQueue()\n", dev_id_in);
#endif
	if(prev_dev_id != dev_id){;
#ifdef UDEBUG
		lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::CommandQueue(): Called for other dev_id = %d\n",
			dev_id, prev_dev_id);
#endif
	}
#ifdef ENABLE_PARALLEL_BACKEND
#ifdef UDEBUG
		lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::CommandQueue(): Initializing parallel queue with %d Backend workers\n",
		dev_id, MAX_BACKEND_L);
#endif
	backend_ctr = 0;
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		cqueue_backend_ptr[par_idx] = malloc(sizeof(hipStream_t));
		hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr[par_idx]);
		massert(hipSuccess == err, "CommandQueue::CommandQueue(%d) - %s\n", dev_id, hipGetErrorString(err));
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);

		cqueue_backend_data[par_idx] = malloc(sizeof(hipblasHandle_t));
		massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate((hipblasHandle_t*) cqueue_backend_data[par_idx]),
			"CommandQueue::CommandQueue(%d): hipblasCreate failed\n", dev_id);
		massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(*((hipblasHandle_t*) cqueue_backend_data[par_idx]), stream),
			"CommandQueue::CommandQueue(%d): hipblasSetStream failed\n", dev_id);
	}
#else
#ifdef UDEBUG
		lprintf(lvl, "[dev_id=%3d] ------- CommandQueue::CommandQueue(%d): Initializing simple queue\n", dev_id);
#endif
	cqueue_backend_ptr = malloc(sizeof(hipStream_t));
	hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr);
	massert(hipSuccess == err, "CommandQueue::CommandQueue(%d) - %s\n", dev_id, hipGetErrorString(err));
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);

	cqueue_backend_data = malloc(sizeof(hipblasHandle_t));
	massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate((hipblasHandle_t*) cqueue_backend_data),
		"CommandQueue::CommandQueue(%d): hipblasCreate failed\n", dev_id);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(*((hipblasHandle_t*) cqueue_backend_data), stream),
		"CommandQueue::CommandQueue(%d): hipblasSetStream failed\n", dev_id);
#endif
	CoCoPeLiaSelectDevice(prev_dev_id);
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::CommandQueue()\n", dev_id);
#endif
}

CommandQueue::~CommandQueue()
{
	#ifdef UDDEBUG
		lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::~CommandQueue()\n", dev_id);
	#endif
		sync_barrier();
		CoCoPeLiaSelectDevice(dev_id);
#ifdef ENABLE_PARALLEL_BACKEND
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);
		hipError_t err = hipStreamSynchronize(stream);
		massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamSynchronize: %s\n", hipGetErrorString(err));
		err = hipStreamDestroy(stream);
		massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
		free(cqueue_backend_ptr[par_idx]);
		hipblasHandle_t handle = *((hipblasHandle_t*) cqueue_backend_data[par_idx]);
		massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(handle),
			"CommandQueue::CommandQueue - hipblasDestroy(handle) failed\n");
	}
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamSynchronize: %s\n", hipGetErrorString(err));
	err = hipStreamDestroy(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
	free(cqueue_backend_ptr);
	hipblasHandle_t handle = *((hipblasHandle_t*) cqueue_backend_data);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(handle),
		"CommandQueue::CommandQueue - hipblasDestroy(handle) failed\n");
#endif
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::~CommandQueue()\n", dev_id);
#endif
	return;
}

void CommandQueue::sync_barrier()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::sync_barrier()\n", dev_id);
#endif
#ifdef ENABLE_PARALLEL_BACKEND
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);
		hipError_t err = hipStreamSynchronize(stream);
		massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
	}
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
#endif
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::sync_barrier()\n", dev_id);
#endif
}

void CommandQueue::add_host_func(void* func, void* data){
	get_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::add_host_func()\n", dev_id);
#endif
#ifdef ENABLE_PARALLEL_BACKEND
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[backend_ctr]);
	hipError_t err = hipLaunchHostFunc(stream, (hipHostFn_t) func, data);
	massert(hipSuccess == err, "CommandQueue::add_host_func - %s\n", hipGetErrorString(err));
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipLaunchHostFunc(stream, (hipHostFn_t) func, data);
	massert(hipSuccess == err, "CommandQueue::add_host_func - %s\n", hipGetErrorString(err));
#endif
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::add_host_func()\n", dev_id);
#endif
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::wait_for_event(Event(%d))\n", dev_id, Wevent->id);
#endif
	if (Wevent->query_status() == CHECKED);
	else{
		// TODO: New addition (?)
		if (Wevent->query_status() == UNRECORDED) error("CommandQueue::wait_for_event:: UNRECORDED event\n");
		get_lock();
#ifdef ENABLE_PARALLEL_BACKEND
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[backend_ctr]);
#else
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
#endif
		hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
		release_lock();
		hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
		massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
	}
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::wait_for_event(Event(%d))\n", dev_id, Wevent->id);
#endif
	return;
}

#ifdef ENABLE_PARALLEL_BACKEND
int CommandQueue::request_parallel_backend()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::request_parallel_backend()\n", dev_id);
#endif
	get_lock();
	if (backend_ctr == MAX_BACKEND_L - 1) backend_ctr = 0;
	else backend_ctr++;
	int tmp_backend_ctr = backend_ctr;
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::request_parallel_backend() = %d\n", dev_id, tmp_backend_ctr);
#endif
	return tmp_backend_ctr;
}

void CommandQueue::set_parallel_backend(int backend_ctr_in)
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> CommandQueue::set_parallel_backend(%d)\n", dev_id, backend_ctr_in);
#endif
	get_lock();
	backend_ctr = backend_ctr_in;
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| CommandQueue::set_parallel_backend(%d)\n", dev_id, backend_ctr);
#endif
	return;
}

#endif

/*****************************************************/
/// Event class functions. TODO: Do status = .. commands need lock?
Event::Event(int dev_id_in)
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::Event()\n", dev_id_in, Event_num_device[idxize(dev_id_in)]);
#endif
	get_lock();
	event_backend_ptr = malloc(sizeof(hipEvent_t));
	id = Event_num_device[idxize(dev_id_in)];
	Event_num_device[idxize(dev_id_in)]++;
#ifndef ENABLE_LAZY_EVENTS
	dev_id = dev_id_in;
	hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
	massert(hipSuccess == err, "Event::Event() - %s\n", hipGetErrorString(err));
#else
	dev_id = dev_id_in - 42;
#endif
	status = UNRECORDED;
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::Event()\n", dev_id, id);
#endif
}

Event::~Event()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::~Event()\n", dev_id, id);
#endif
	sync_barrier();
	get_lock();
#ifndef ENABLE_LAZY_EVENTS
	Event_num_device[idxize(dev_id)]--;
	hipError_t err = hipEventDestroy(*(( hipEvent_t*) event_backend_ptr));
	massert(hipSuccess == err, "Event(%d)::~Event() - %s\n", id, hipGetErrorString(err));
#else
	if (dev_id < -1) 	Event_num_device[idxize(dev_id+42)]--;
	else{
			Event_num_device[idxize(dev_id)]--;
			hipError_t err = hipEventDestroy(*(( hipEvent_t*) event_backend_ptr));
			massert(hipSuccess == err, "Event(%d)::~Event() - %s\n", id, hipGetErrorString(err));
	}
#endif
	free(event_backend_ptr);
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::~Event()\n", dev_id, id);
#endif
}

void Event::sync_barrier()
{
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::sync_barrier()\n", dev_id, id);
#endif
	//get_lock();
	if (status != CHECKED){
		if (status == UNRECORDED){;
#ifdef UDEBUG
			warning("[dev_id=%3d] |-----> Event(%d)::sync_barrier() - Tried to sync unrecorded event\n", dev_id, id);
#endif
		}
		else{
			hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
			hipError_t err = hipEventSynchronize(cuda_event);
			if (status == RECORDED) status = CHECKED;
			massert(hipSuccess == err, "Event::sync_barrier() - %s\n", hipGetErrorString(err));
		}
	}
	//release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::sync_barrier()\n", dev_id, id);
#endif
	return;
}

void Event::record_to_queue(CQueue_p Rr){
	get_lock();
	if (Rr == NULL){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----> Event(%d)::record_to_queue(NULL)\n", dev_id, id);
#endif
		status = CHECKED;
		release_lock();
		return;
	}
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::record_to_queue(Queue(dev_id=%d))\n", dev_id, id, Rr->dev_id);
#endif
	int prev_dev_id;
	hipGetDevice(&prev_dev_id);
	if (Rr->dev_id != prev_dev_id){
		CoCoPeLiaSelectDevice(Rr->dev_id);
#ifdef UDEBUG
		warning("Event(%d,dev_id = %d)::record_to_queue(%d): caller prev_dev_id=%d, changing to %d\n",
		id, dev_id, Rr->dev_id, prev_dev_id, Rr->dev_id);
#endif
	}
	if (status != UNRECORDED){
		;
#ifdef UDEBUG
		warning("Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event\n",
			id, dev_id, Rr->dev_id, print_event_status(status));
#endif
#ifdef ENABLE_LAZY_EVENTS
		if(Rr->dev_id != dev_id)
			error("(Lazy)Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event in iligal dev\n",
				id, dev_id, Rr->dev_id, print_event_status(status));
#endif
	}
#ifdef ENABLE_LAZY_EVENTS
	else if (status == UNRECORDED){
		if(dev_id > -1) /// TODO: This used to be an error, but with soft reset it was problematic...is it ok?
			;//warning("(Lazy)Event(%d,dev_id = %d)::record_to_queue(%d) - UNRECORDED event suspicious dev_id\n",
			//	id, dev_id, Rr->dev_id);
		dev_id = Rr->dev_id;
		hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
		massert(hipSuccess == err, "(Lazy)Event(%d,dev_id = %d)::record_to_queue(%d): - %s\n",
			id, dev_id, Rr->dev_id, hipGetErrorString(err));
	}
#endif
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
#ifdef ENABLE_PARALLEL_BACKEND
	hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr[Rr->backend_ctr]);
	hipError_t err = hipEventRecord(cuda_event, stream);
#else
	hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr);
	hipError_t err = hipEventRecord(cuda_event, stream);
#endif
	status = RECORDED;
	massert(hipSuccess == err, "Event(%d,dev_id = %d)::record_to_queue(%d) - %s\n",  id, dev_id, Rr->dev_id, hipGetErrorString(err));
	if (Rr->dev_id != prev_dev_id){
		hipSetDevice(prev_dev_id);
	}
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::record_to_queue(Queue(dev_id=%d))\n", dev_id, id, Rr->dev_id);
#endif
}

event_status Event::query_status(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::query_status()\n", dev_id, id);
#endif
	get_lock();
	enum event_status local_status = status;
	if (local_status != CHECKED){
#ifdef ENABLE_LAZY_EVENTS
		if (local_status == UNRECORDED){
			release_lock();
			return UNRECORDED;
		}
#endif
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipError_t err = hipEventQuery(cuda_event);

		if (err == hipSuccess && (local_status == UNRECORDED ||  local_status == COMPLETE));
		else if (err == hipSuccess && local_status == RECORDED) local_status = status = COMPLETE;
		else if (err == hipErrorNotReady && local_status == RECORDED);
		else if (err == hipErrorNotReady && local_status == UNRECORDED){
#ifdef UDEBUG
			// this should not happen in a healthy locked update scenario.
			warning("Event::query_status(): hipErrorNotReady with status == UNRECORDED should not happen\n");
#endif
			local_status = status = RECORDED;
		}
		else if (err == hipSuccess &&  local_status == CHECKED){
			;
			// TODO: This should not happen in a healthy locked update scenario.
			// But it does since no locking yet. Not sure of its effects.
#ifdef UDEBUG
			warning("[dev_id=%3d] |-----> Event(%d)::query_status(): hipSuccess with local_status == CHECKED should not happen\n", dev_id, id);
#endif
		}
		else error("[dev_id=%3d] |-----> Event(%d)::query_status() - %s, local_status=%s, status = %s\n", dev_id, id,
		hipGetErrorString(err), print_event_status(local_status), print_event_status(status));
	}
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::query_status() = %s\n", dev_id, id, print_event_status(status));
#endif
	return local_status;
}

void Event::checked(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::checked()\n", dev_id, id);
#endif
	get_lock();
	if (status == COMPLETE) status = CHECKED;
	else error("Event::checked(): error event was %s,  not COMPLETE()\n", print_event_status(status));
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::checked()\n", dev_id, id);
#endif
}

void Event::soft_reset(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::soft_reset()\n", dev_id, id);
#endif
	//sync_barrier();
	get_lock();
	//event_status prev_status = status;
	status = UNRECORDED;
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::soft_reset()\n", dev_id, id);
#endif
}

void Event::reset(){
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] |-----> Event(%d)::reset()\n", dev_id, id);
#endif
	sync_barrier();
	get_lock();
	event_status prev_status = status;
	status = UNRECORDED;
#ifdef ENABLE_LAZY_EVENTS
	if(dev_id >= -1){
		dev_id = dev_id - 42;
		hipError_t err = hipEventDestroy(*(( hipEvent_t*) event_backend_ptr));
		massert(hipSuccess == err, "[dev_id=%3d] (Lazy)Event(%d)::reset - %s\n", dev_id + 42, id, hipGetErrorString(err));
	}
#endif
	release_lock();
#ifdef UDDEBUG
	lprintf(lvl, "[dev_id=%3d] <-----| Event(%d)::reset()\n", dev_id, id);
#endif
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer(int dev_id) {
  Event_start = new Event(dev_id);
  Event_stop = new Event(dev_id);
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
}

double Event_timer::sync_get_time()
{
	float temp_t;
	if(Event_start->query_status() != UNRECORDED){
		Event_start->sync_barrier();
		if(Event_stop->query_status() != UNRECORDED) Event_stop->sync_barrier();
		else error("Event_timer::sync_get_time: Event_start is %s but Event_stop still UNRECORDED\n",
			print_event_status(Event_start->query_status()));
		hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
		hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
		hipEventElapsedTime(&temp_t, cuda_event_start, cuda_event_stop);
	}
	else temp_t = 0;
	time_ms = (double) temp_t;
	return time_ms;
}

/*****************************************************/
