#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A hipblasSgemm micro-benchmark
///

#include <cassert>
#include "microbenchmarks.hpp"
//TODO: This should at some point be removed (some fuctions require wrapping)
#include "backend_wrappers.hpp"

void report_run(char* filename, size_t M, size_t N, size_t K, double mean_t, double margin_err, size_t sample_sz, double bench_t){

	FILE* fp = fopen(filename,"a");
	if (!fp) error("report_run: LogFile failed to open");
   	fprintf(fp,"%d,%d,%d, %e,%e,%zu,%e\n", M, N, K, mean_t, margin_err, sample_sz, bench_t);
        fclose(fp); 
}

int main(const int argc, const char *argv[]) {

  	float alpha, beta;
  	alpha = 1.1234, beta = 1.2345;

	char TransA, TransB; 
  	int ctr = 1, dev_id;

	switch (argc) {
	case (4):
		dev_id = atoi(argv[ctr++]);
		TransA = argv[ctr++][0];
		TransB = argv[ctr++][0]; 
		break;
	default:
		error("Incorrect input arguments. Usage: ./correct_run dev_id TransA TransB\n");
  	}

	// Define the max size of a benchmark kernel to run on this machine. 
	size_t maxDim = CoCoGetMaxDimSqAsset2D(3, sizeof(float), STEP_TRANS, dev_id);

	char *filename = (char *) malloc(256* sizeof(char));
	sprintf(filename, "%s/Benchmark-Results/cublasSgemm_dev-%d_TransA-%c_TransB-%c_%s.log", DEPLOYDB, dev_id, TransA, TransB, VERSION);
	check_benchmark(filename);

	size_t ldA = maxDim, ldB = maxDim, ldC = maxDim;

	/// Matrix Layouts for CPU GEMM
	CBLAS_TRANSPOSE dummy1;    // CblasNoTrans, CblasTrans
	hipblasOperation_t gpu_op_A, gpu_op_B; // HIPBLAS_OP_N, HIPBLAS_OP_T
	
	TransposeTranslate(TransA, &dummy1, &gpu_op_A, &ldA, maxDim, maxDim);
	TransposeTranslate(TransB, &dummy1, &gpu_op_B, &ldB, maxDim, maxDim);

	/// Set device 
	hipSetDevice(dev_id);

	hipblasHandle_t handle0;
 	hipStream_t host_stream;

  	hipStreamCreate(&host_stream);
	assert(HIPBLAS_STATUS_SUCCESS == hipblasCreate(&handle0));
	assert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(handle0, host_stream));

	fprintf(stderr, "\nAllocating device memory...");
	double cpu_timer = csecond();

	float *A_dev, *B_dev, *C_dev;
  	A_dev = (float*) CoCoMalloc(maxDim * maxDim * sizeof(float), dev_id);
  	B_dev = (float*) CoCoMalloc(maxDim * maxDim * sizeof(float), dev_id);
  	C_dev = (float*) CoCoMalloc(maxDim * maxDim * sizeof(float), dev_id);
	CoCoSyncCheckErr();

	cpu_timer  = csecond() - cpu_timer ;
	fprintf(stderr, "done.\nAlloc time:\t%lf ms\n\n",  cpu_timer  * 1000);

	fprintf(stderr, "Initializing to random values..."); 
	cpu_timer = csecond();

	CoCoVecInit(A_dev, maxDim * maxDim, 42, dev_id);
	CoCoVecInit(B_dev, maxDim * maxDim, 42, dev_id);
	CoCoVecInit(C_dev, maxDim * maxDim, 42, dev_id);

	CoCoSyncCheckErr();
	cpu_timer  = csecond() - cpu_timer ;	
	fprintf(stderr, "done.\nInit time:\t%lf ms\n\n",  cpu_timer  * 1000);

	fprintf(stderr, "\nMatrix details: A(%s) B(%s) C(%s) -> Mmax = %d, Nmax = %d, Kmax = %d\n",
            print_mem(COL_MAJOR), print_mem(COL_MAJOR), print_mem(COL_MAJOR), maxDim, maxDim, maxDim);
	fprintf(stderr, "Constants: alpha = %lf, beta = %lf\n", alpha, beta);

	// Warmup 
	for ( int itt = 0; itt <1; itt++){
		assert(HIPBLAS_STATUS_SUCCESS == hipblasSgemm(handle0, gpu_op_A, gpu_op_B, maxDim, maxDim, maxDim, &alpha, A_dev, ldA, B_dev, ldB, &beta, C_dev, ldC));
		hipStreamSynchronize(host_stream);
	}
	CoCoSyncCheckErr();
#ifdef AUTO_BENCH_USE_BOOST
	double cublas_t_vals[MICRO_MAX_ITER], cublas_t_sum, cublas_t_mean, bench_t, error_margin; 
	size_t bench_ctr = 0, sample_sz, step = STEP_BLAS3;
	for (size_t T = MIN_DIM_BLAS3; T < maxDim + 1; T+=step){
		if (T >= step * 16) step *=2; 
		fprintf(stderr,"Running hipblasSgemm-> square T = %d:\n", T);
		cublas_t_mean = cublas_t_sum = error_margin = 0;
		sample_sz = 0; 
		bench_t = csecond();
		double std_dev = 0; 
		for (sample_sz = 1; sample_sz < MICRO_MAX_ITER + 1; sample_sz++) {	
			cpu_timer = csecond();
			assert(HIPBLAS_STATUS_SUCCESS == hipblasSgemm(handle0, gpu_op_A, gpu_op_B, T, T, T, &alpha, A_dev, ldA, B_dev, ldB, &beta, C_dev, ldC));
			hipStreamSynchronize(host_stream);
			cpu_timer  = csecond() - cpu_timer ;
			cublas_t_vals[sample_sz-1] = cpu_timer;
			cublas_t_sum += cublas_t_vals[sample_sz-1];
			cublas_t_mean = cublas_t_sum/sample_sz; 
			if (sample_sz < 2) continue;
			for (int i = 0; i < sample_sz; i++) std_dev += pow(cublas_t_vals[i] - cublas_t_mean, 2);
			std_dev /= sample_sz;
    			std_dev = sqrt(std_dev);
			boost::math::students_t dist(sample_sz - 1);
			double Td = boost::math::quantile(boost::math::complement(dist, alphaCI / 2));
			error_margin = Td*std_dev/sqrt(sample_sz);
			//fprintf(stderr, "\tItter %d:\t mean=%lf, std_dev = %lf, Error margin =%lf\n", sample_sz, cublas_t_mean , std_dev, error_margin);
			if (sample_sz > MICRO_MIN_ITER && error_margin/cublas_t_mean  * 100 <= 5) break; 
		}
		bench_t = csecond() - bench_t;
		fprintf(stderr, "Microbenchmark (M = N = K = %zu) complete:\t mean_exec_t=%lf ms ( %.1lf Gflops/s ), Error Margin (percentage of mean) = %lf %, Itter = %d, Microbench_t = %lf\n\n", T, cublas_t_mean  * 1000, Gval_per_s(dgemm_flops(T,T,T), cublas_t_mean), error_margin/cublas_t_mean  * 100, sample_sz, bench_t);
		CoCoSyncCheckErr();

		report_run(filename, T, T, T, cublas_t_mean, error_margin, sample_sz, bench_t); 
		bench_ctr++;
	}
#else
	double  bench_t, cublas_t_av, cublas_t_min , cublas_t_max; 
	size_t bench_ctr = 0, step = STEP_BLAS3;
	for (size_t T = MIN_DIM_BLAS3; T < maxDim + 1; T+=step){
		if (T >= step * 16) step *=2; 
		fprintf(stderr,"Running hipblasSgemm-> square T = %d:\n", T);
		cublas_t_av = cublas_t_max = 0;
		cublas_t_min = 1e9;
		bench_t = csecond();
		for (int itt = 0; itt < ITER; itt ++) {
			cpu_timer = csecond();
			assert(HIPBLAS_STATUS_SUCCESS == hipblasSgemm(handle0, gpu_op_A, gpu_op_B, T, T, T, &alpha, A_dev, ldA, B_dev, ldB, &beta, C_dev, ldC));
			hipStreamSynchronize(host_stream);
			cpu_timer  = csecond() - cpu_timer ;
			cublas_t_av += cpu_timer;
			if (cpu_timer > cublas_t_max) cublas_t_max = cpu_timer; 
			if (cpu_timer < cublas_t_min) cublas_t_min = cpu_timer; 
		}
		bench_t = csecond() - bench_t;
		cublas_t_av /= ITER;
		fprintf(stderr, "GPU exec time:\t Average=%lf ms, Min = %lf ms, Max = %lf ms\n", cublas_t_av  * 1000, cublas_t_min  * 1000, cublas_t_max  * 1000);
		CoCoSyncCheckErr();

		report_run(filename, T, T, T, cublas_t_av, fmax(cublas_t_max - cublas_t_av, cublas_t_av - cublas_t_min), ITER, bench_t); 
		bench_ctr++;
	}
#endif
	fprintf(stderr, "Ran %d Benchmarks.Finallizing...\n", bench_ctr);
	return 0;
}
