#include "hip/hip_runtime.h"
///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief A transfer micro-benchmark from->to for a) contiguous transfers, b) non-cont square transfers, c) full bidirectional overlapped transfers
///

#include <unistd.h>
#include <cassert>
#include "microbenchmarks.hpp"

/// TODO: This is the number of reverse-link transfers used for overlaping.
/// The benchmark results are correct ONLY if D2H_BW < MAX_ASSUMED_H2D_TIMES_SLOWER * H2D_BW.
/// For current systems 2-4 is sufficient - larger multipliers increase total benchmark time proportionally.
#define MAX_ASSUMED_H2D_TIMES_SLOWER 2

void report_run(char* filename, long int dim_1, long int dim_2, double mean_t, double margin_err, long int sample_sz, double mean_t_bid, double margin_err_bid, long int sample_sz_bid, double bench_t){

	FILE* fp = fopen(filename,"a");
	if (!fp) error("report_run: LogFile failed to open");
   	fprintf(fp,"%d,%d, %e,%e,%zu, %e,%e,%zu, %e\n", dim_1, dim_2, mean_t, margin_err, sample_sz, mean_t_bid, margin_err_bid, sample_sz_bid, bench_t);
        fclose(fp);
}

int main(const int argc, const char *argv[]) {

  int ctr = 1, samples, dev_id, dev_count;

	short from, to;
	long int minDim = MIN_DIM_TRANS, maxDim = 0, step = STEP_TRANS;

	switch (argc) {
	case (3):
		to = atoi(argv[ctr++]);
		from = atoi(argv[ctr++]);
		break;
	default:
		error("Incorrect input arguments. Usage: ./correct_run to from\n");
  	}

	if (from == to) error("Transfer benchmark@%s %d->%d: Same device\n",TESTBED, from, to);

	char *filename = (char *) malloc(256* sizeof(char));
	sprintf(filename, "%s/Benchmark-Results/CoCoMemcpy2DAsync_to-%d_from-%d_%s.log", DEPLOYDB, to, from, VERSION);
	check_benchmark(filename);

	// Define the max size of a benchmark kernel to run on this machine.
	maxDim = min(CoCoGetMaxDimSqAsset2D(3, sizeof(double), STEP_TRANS, to),
	min(CoCoGetMaxDimSqAsset2D(3, sizeof(double), STEP_TRANS, from), MAX_DIM_TRANS)) ;

	fprintf(stderr,"\nTransfer benchmark@%s %d->%d : (%d,%d) with step %d\n", TESTBED, from, to, minDim, maxDim, step);

	hipGetDeviceCount(&dev_count);

	if (minDim < 1) error("Transfer Microbench: Bytes must be > 0");
	else if ( dev_count < from + 1) error("Transfer Microbench: Src device does not exist");
	else if ( dev_count < to + 1) error("Transfer Microbench: Dest device does not exist");

	void* src, *dest, *rev_src, *rev_dest;

	//Only model pinned memory transfers from host to dev and visa versa
  	if (from < 0 && to < 0) error("Transfer Microbench: Both locations are in host");
	else if (from == -2 || to == -2) error("Transfer Microbench: Not pinned memory (synchronous)");
  	else if ( from >= 0 && to >= 0){
		short dev_id[2], num_devices = 2;
		dev_id[0] = from;
		dev_id[1] = to;
		// Check/Enable peer access between participating GPUs
		CoCoEnableLinks(0, num_devices);
		// Check/Enable peer access between participating GPUs
		CoCoEnableLinks(1, num_devices);
	}
	else if(from >= 0) hipSetDevice(from);
	else if(to >= 0) hipSetDevice(to);

	long int ldsrc, ldest = ldsrc = maxDim + 1;

	src = CoCoMalloc(maxDim*(maxDim+1)*sizeof(double), from);
	dest =  CoCoMalloc(maxDim*(maxDim+1)*sizeof(double), to);
	rev_src = CoCoMalloc(maxDim*(maxDim+1)*sizeof(double), to);
	rev_dest = CoCoMalloc(maxDim*(maxDim+1)*sizeof(double), from);

	CoCoVecInit((double*)src, maxDim*(maxDim+1), 42, from);
	CoCoVecInit((double*)rev_src, maxDim*(maxDim+1), 43, to);

	CQueue_p transfer_link = new CommandQueue(from), reverse_link = new CommandQueue(from);

	CoCoSyncCheckErr();
	fprintf(stderr, "Warming up...\n");
	/// Warmup.
	for (int it = 0; it < 10; it++) CoCoMemcpy2DAsync(dest, ldest, src, ldsrc, maxDim, maxDim, sizeof(double), to, from, transfer_link);
	CoCoSyncCheckErr();
#ifdef AUTO_BENCH_USE_BOOST
	double cpu_timer, transfer_t_vals[MICRO_MAX_ITER], transfer_t_sum, transfer_t_mean, bench_t, error_margin;
	double transfer_t_bid_sum, transfer_t_bid_mean, error_margin_bid;
	long int sample_sz, sample_sz_bid;
	CoCoPeLiaSelectDevice(from);
	Event_timer_p device_timer = new Event_timer(from);
	for (long int dim = minDim; dim < MAX_DIM_BLAS3; dim+=step){ // maxDim+1
		if (dim >= step * 16) step*=2;
		transfer_t_sum = transfer_t_mean = bench_t = error_margin = 0;
		fprintf(stderr, "Cublas-chunk Link %d->%d (Chunk %dx%d):\n", from, to, dim, dim);
		sample_sz = 0;
		bench_t = csecond();
		double std_dev = 0;
		for (sample_sz = 1; sample_sz < MICRO_MAX_ITER + 1; sample_sz++) {
			cpu_timer = csecond();
			CoCoMemcpy2DAsync(dest, ldest, src, ldsrc, dim, dim, sizeof(double), to, from, transfer_link);
			transfer_link->sync_barrier();
			cpu_timer  = csecond() - cpu_timer ;
			transfer_t_vals[sample_sz-1] = cpu_timer;
			transfer_t_sum += transfer_t_vals[sample_sz-1];
			transfer_t_mean = transfer_t_sum/sample_sz;
			if (sample_sz < 2) continue;
			for (int i = 0; i < sample_sz; i++) std_dev += pow(transfer_t_vals[i] - transfer_t_mean, 2);
			std_dev /= sample_sz;
    			std_dev = sqrt(std_dev);
			boost::math::students_t dist(sample_sz - 1);
			double Td = boost::math::quantile(boost::math::complement(dist, alphaCI / 2));
			error_margin = Td*std_dev/sqrt(sample_sz);
			//fprintf(stderr, "\tItter %d:\t mean=%lf, std_dev = %lf, Error margin =%lf\n", sample_sz, cublas_t_mean , std_dev, error_margin);
			if (sample_sz > MICRO_MIN_ITER && error_margin/transfer_t_mean  * 100 <= 5) break;
		}
		bench_t = csecond() - bench_t;
		fprintf(stderr, "Microbenchmark (dim1 = dim2 = %zu) complete:\t mean_exec_t=%lf ms  ( %lf Gb/s), Error Margin (percentage of mean) = %lf %, Itter = %d, Microbench_t = %lf\n\n", dim, transfer_t_mean  * 1000, Gval_per_s(dim*dim*8, transfer_t_mean), error_margin/transfer_t_mean  * 100, sample_sz, bench_t);
		CoCoSyncCheckErr();

		transfer_t_bid_sum = transfer_t_bid_mean = error_margin_bid = 0;
		fprintf(stderr, "Reverse overlapped Link %d->%d (Chunk %dx%d):\n", from, to, dim, dim);
		sample_sz_bid = 0;
		bench_t = csecond() - bench_t;
		std_dev = 0;
		for (sample_sz_bid = 1; sample_sz_bid < MICRO_MAX_ITER + 1; sample_sz_bid++) {
			for (int rep = 0; rep < MAX_ASSUMED_H2D_TIMES_SLOWER; rep++) CoCoMemcpy2DAsync(rev_dest, ldest, rev_src, ldsrc, dim, dim, sizeof(double), from, to, reverse_link);
			device_timer->start_point(transfer_link);
			CoCoMemcpy2DAsync(dest, ldest, src, ldsrc, dim, dim, sizeof(double), to, from, transfer_link);
			device_timer->stop_point(transfer_link);
			CoCoSyncCheckErr();
			transfer_t_vals[sample_sz_bid-1] = device_timer->sync_get_time()/1000;
			transfer_t_bid_sum += transfer_t_vals[sample_sz_bid-1];
			transfer_t_bid_mean = transfer_t_bid_sum/sample_sz_bid;
			if (sample_sz_bid < 2) continue;
			for (int i = 0; i < sample_sz_bid; i++) std_dev += pow(transfer_t_vals[i] - transfer_t_bid_mean, 2);
			std_dev /= sample_sz_bid;
    			std_dev = sqrt(std_dev);
			boost::math::students_t dist(sample_sz_bid - 1);
			double Td = boost::math::quantile(boost::math::complement(dist, alphaCI / 2));
			error_margin_bid = Td*std_dev/sqrt(sample_sz_bid);
			//fprintf(stderr, "\tItter %d:\t mean=%lf, std_dev = %lf, Error margin =%lf\n", sample_sz_bid, cublas_t_mean , std_dev, error_margin_bid);
			if (sample_sz_bid > MICRO_MIN_ITER && error_margin_bid/transfer_t_bid_mean  * 100 <= 5) break;
		}
		bench_t = csecond() - bench_t;
		fprintf(stderr, "Microbenchmark (dim1 = dim2 = %zu) complete:\t mean_exec_t=%lf ms  ( %lf Gb/s), Error Margin (percentage of mean) = %lf %, Itter = %d, Microbench_t = %lf\n\n", dim, transfer_t_bid_mean  * 1000, Gval_per_s(dim*dim*8, transfer_t_bid_mean), error_margin_bid/transfer_t_bid_mean  * 100, sample_sz_bid, bench_t);
		CoCoSyncCheckErr();

		report_run(filename, dim, dim , transfer_t_mean, error_margin, sample_sz, transfer_t_bid_mean, error_margin_bid, sample_sz_bid, bench_t);
	}
#else
	/// Local Timers
	double cpu_timer, t_sq_av, t_sq_min, t_sq_max, t_sq_bid_av, t_sq_bid_min, t_sq_bid_max, bench_t;
	Event_timer_p device_timer = new Event_timer();
	for (long int dim = minDim; dim < maxDim+1; dim+=step){
		if (dim >= step * 16) step*=2;
		t_sq_av = t_sq_max = t_sq_bid_av = t_sq_bid_max = bench_t= 0;
		t_sq_min = t_sq_bid_min = 1e9;
		fprintf(stderr, "Cublas-chunk Link %d->%d (Chunk %dx%d):\n", from, to, dim, dim);
		bench_t = csecond();
		for (int it = 0; it < ITER ; it++) {
			cpu_timer = - csecond();
			CoCoMemcpy2DAsync(dest, ldest, src, ldsrc, dim, dim, sizeof(double), to, from, transfer_link);
			transfer_link->sync_barrier();
			cpu_timer = csecond() + cpu_timer;
			t_sq_av += cpu_timer;
			if (cpu_timer > t_sq_max) t_sq_max = cpu_timer;
			if (cpu_timer < t_sq_min) t_sq_min = cpu_timer;
		}
		CoCoSyncCheckErr();
		t_sq_av = t_sq_av/ITER;
		fprintf(stderr, "Transfer time:\t Average=%lf ms ( %lf Gb/s), Min = %lf ms, Max = %lf ms\n", t_sq_av  * 1000, Gval_per_s(dim*dim*8, t_sq_av), t_sq_min  * 1000, t_sq_max  * 1000);

		fprintf(stderr, "Reverse overlapped Link %d->%d (Chunk %dx%d):\n", from, to, dim, dim);
		for (int it = 0; it < ITER ; it++) {
			for (int rep = 0; rep < MAX_ASSUMED_H2D_TIMES_SLOWER ; rep++) CoCoMemcpy2DAsync(rev_dest, ldest, rev_src, ldsrc, dim, dim, sizeof(double), from, to, reverse_link);
			device_timer->start_point(transfer_link);
			CoCoMemcpy2DAsync(dest, ldest, src, ldsrc, dim, dim, sizeof(double), to, from, transfer_link);
			device_timer->stop_point(transfer_link);
			CoCoSyncCheckErr();
			t_sq_bid_av += device_timer->sync_get_time();
			if (device_timer->sync_get_time() > t_sq_bid_max) t_sq_bid_max = device_timer->sync_get_time();
			if (device_timer->sync_get_time() < t_sq_bid_min) t_sq_bid_min = device_timer->sync_get_time();
		}
		CoCoSyncCheckErr();
		t_sq_bid_av = t_sq_bid_av/ITER/1000;
		t_sq_bid_min/= 1000;
		t_sq_bid_max/= 1000;
		bench_t = csecond() - bench_t;
		fprintf(stderr, "Transfer time:\t Average=%lf ms ( %lf Gb/s), Min = %lf ms, Max = %lf ms\n", t_sq_bid_av  * 1000, Gval_per_s(dim*dim*8, t_sq_bid_av), t_sq_bid_min  * 1000, t_sq_bid_max  * 1000);
		report_run(filename, dim, dim, t_sq_av, fmax(t_sq_max - t_sq_av, t_sq_av - t_sq_min), ITER, t_sq_bid_av, fmax(t_sq_bid_max - t_sq_bid_av, t_sq_bid_av - t_sq_bid_min), ITER, bench_t);

	}
#endif
	CoCoFree(&src, from);
	CoCoFree(&dest, to);
	CoCoFree(&rev_src, to);
	CoCoFree(&rev_dest, from);
	return 0;
}
